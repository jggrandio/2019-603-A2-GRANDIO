#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <float.h>
#include <math.h>
#include <iostream>
#include "libarff/arff_parser.h"
#include "libarff/arff_data.h"
using namespace std;

__global__ void sKNN(float *attr, int *val, int n_att, int n_inst, float *distance, int com, float *smallestDistance){

	//calculate tid
	extern __shared__ float sh[];
	int column = (blockDim.x * blockIdx.x) + threadIdx.x;
	int row = (blockDim.y * blockIdx.y) + threadIdx.y;
	int tid = (blockDim.x*gridDim.x*row)+column;

	if (tid < n_inst * com){
		smallestDistance[tid] = FLT_MAX;
	}

		if (column < n_inst * n_att && row < n_inst){
			if(threadIdx.x == 0){
				distance[row * n_inst + blockIdx.x] = 0; //Distance to 0 so the first one is a min distance
			}

			float diff;

			diff = (attr[row* n_att + threadIdx.x] - attr[column])*(attr[row* n_att + threadIdx.x] - attr[column]);
			sh[threadIdx.y*n_att+threadIdx.x]=diff;
			__syncthreads();

			if(threadIdx.x == 0){

				for (int k = threadIdx.y*n_att; k < (threadIdx.y*n_att) + n_att; k++){


					distance[row * n_inst + blockIdx.x] += sh[k];

				}

				distance[row * n_inst + blockIdx.x] =  sqrt(distance[row * n_inst + blockIdx.x]); //Distance to 0 so the first one is a min distance


				if (row == blockIdx.x){ // when it is the same point
					distance[row * n_inst + blockIdx.x] = FLT_MAX;
				}
			}

			//for(int a = 0; a<n_inst; a++){
			//	for(int b = 0; b<n_inst; b++){
			//		if (row == a && column == b){
			//			printf("element (%d, %d): %f \n",a, b, distance[row * n_inst + column/n_att]);
			//		}
			//	}
			//}
		}
}

__global__ void pred(int *pred, int com, int n_inst, float *distance, float *smallestDistance, int* smallestDistanceClass, int *val){
	int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
	if (tid < n_inst){
		for(int j = 0; j < n_inst; j++){
			for (int n = tid * com ; n<tid * com + com; n++)
				{
					if(distance[n_inst*j+tid] < smallestDistance[n]) // select the closest one
					{
						for (int t=tid * com + com-1; t>n; t--)
						{
							smallestDistance[t] = smallestDistance[t-1];
							smallestDistanceClass[t] = smallestDistanceClass[t-1];
						}
						smallestDistance[n] = distance[n_inst*j+tid];
						smallestDistanceClass[n] = val[j];
						break;
					}
				}
		}
		int freq = 0;
		int predict=0;
		for ( int m = tid * com; m<tid * com + com; m++)
		{
			int tfreq = 1;
			int tpredict=smallestDistanceClass[m];
			for (int s = m+1 ; s< tid * com + com; s++)
			{
				if (tpredict==smallestDistanceClass[s])
				{
					tfreq++;
				}
			}
			if (tfreq>freq)
			{
				predict=smallestDistanceClass[m];
				freq=tfreq;
			}
		}

		pred[tid]= predict;
	}
}

int* KNN(ArffData* dataset, int com)
{
	int n_att = dataset->num_attributes() - 1;
	int n_inst = dataset->num_instances();

	int threadperblockdim = n_att;
	int griddimx = ((n_att*n_inst) + threadperblockdim - 1) / threadperblockdim;
	int griddimy = (n_inst + threadperblockdim - 1) / threadperblockdim;

	dim3 blocksize(threadperblockdim,threadperblockdim);
	dim3 gridsize(griddimx,griddimy);


	int *h_pred= (int*)malloc(n_inst * sizeof(int));
	int *h_val= (int*)malloc(n_inst * sizeof(int));
	float *h_at= (float*)malloc(n_inst * n_att * sizeof(float));


	float *d_at, *d_dist;
	int *d_val, *d_pred;
	hipMalloc(&d_at, n_inst * n_att * sizeof(float));
	hipMalloc(&d_dist, n_inst * n_inst * sizeof(float));
	hipMalloc(&d_val, n_inst* sizeof(int));
	hipMalloc(&d_pred, n_inst* sizeof(int));

	float* smallestDistance;
	int* smallestDistanceClass;
	hipMalloc(&smallestDistance,n_inst * com * sizeof(float));
	hipMalloc(&smallestDistanceClass,n_inst * com * sizeof(int));

	for (int i = 0; i<n_inst; i++){
		h_val[i] = dataset->get_instance(i)->get(dataset->num_attributes() - 1)->operator int32();

		for( int k = 0; k < n_att; k++){
			h_at[i*n_att+k] = dataset->get_instance(i)->get(k)->operator float();
		}
	}
	hipMemcpy(d_val,h_val, n_inst* sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_at,h_at, n_att * n_inst* sizeof(float), hipMemcpyHostToDevice);
	sKNN<<<gridsize , blocksize,n_att*n_att*sizeof(float)>>>(d_at, d_val, n_att, n_inst, d_dist, com, smallestDistance);


	int threadperblock = 256;
	int blocks = (dataset->num_instances() + threadperblock - 1) / threadperblock;
	pred<<<blocks , threadperblock>>>(d_pred, com, n_inst, d_dist, smallestDistance, smallestDistanceClass, d_val);

	hipMemcpy(h_pred, d_pred, n_inst* sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_at);
	hipFree(d_dist);
	hipFree(d_val);
	hipFree(d_pred);
	hipFree(smallestDistance);
	hipFree(smallestDistanceClass);

	return h_pred;
}


int* computeConfusionMatrix(int* predictions, ArffData* dataset)
{
    int* confusionMatrix = (int*)calloc(dataset->num_classes() * dataset->num_classes(), sizeof(int)); // matriz size numberClasses x numberClasses

    for(int i = 0; i < dataset->num_instances(); i++) // for each instance compare the true class and predicted class
    {
        int trueClass = dataset->get_instance(i)->get(dataset->num_attributes() - 1)->operator int32();
        int predictedClass = predictions[i];

        confusionMatrix[trueClass*dataset->num_classes() + predictedClass]++;
    }

    return confusionMatrix;
}

float computeAccuracy(int* confusionMatrix, ArffData* dataset)
{
    int successfulPredictions = 0;

    for(int i = 0; i < dataset->num_classes(); i++)
    {
        successfulPredictions += confusionMatrix[i*dataset->num_classes() + i]; // elements in the diagonal are correct predictions
    }

    return successfulPredictions / (float) dataset->num_instances();
}

int main(int argc, char *argv[])
{
    if(argc != 3)
    {
        cout << "Usage: ./main datasets/datasetFile.arff" << endl;
		cout << "Usage: k value" << endl;
        exit(0);
    }

    ArffParser parser(argv[1]);
    ArffData *dataset = parser.parse();
    struct timespec start, end;

	int k;

	sscanf(argv[2], "%d", &k);


	clock_gettime(CLOCK_MONOTONIC_RAW, &start);

	int* predictions = KNN(dataset,k);

    clock_gettime(CLOCK_MONOTONIC_RAW, &end);
    uint64_t diff = (1000000000L * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec) / 1e6;

    int* confusionMatrix = computeConfusionMatrix(predictions, dataset);
    float accuracy = computeAccuracy(confusionMatrix, dataset);

    printf("The KNN classifier  for %lu instances required %llu ms CPU time, accuracy was %.4f\n", dataset->num_instances(), (long long unsigned int) diff, accuracy);
}
